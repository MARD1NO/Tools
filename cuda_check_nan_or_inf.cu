
#include <hip/hip_runtime.h>
constexpr int LAUNCH_CHECK_NAN_BLOCK_SIZE = 1024; 
constexpr int LAUNCH_CHECK_NAN_GRID_SIZE = 12; 

#define FINAL_MASK 0xffffffff

template <typename T>
__inline__ __device__ T warpReduceSum(T val) {
#pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  return val;
}

template <typename T>
__inline__ __device__ T blockReduceSum(T val) {
  static __shared__ T shared[32];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;

  val = warpReduceSum<T>(val);

  if (lane == 0) shared[wid] = val;

  __syncthreads();

  // Modify from blockDim.x << 5 to blockDim.x / 32. to prevent
  // blockDim.x is not divided by 32
  val = (threadIdx.x < (blockDim.x / 32)) ? shared[lane] : (T)(0.0f);
  val = warpReduceSum<T>(val);

  return val;
}

__global__ void reset_nan_inf_ptr(int32_t* block_num_nan_ptr,
                                  int32_t* block_num_inf_ptr, 
                                  const int32_t elem_cnt){
  for(int i = 0; i < elem_cnt; i++){
    block_num_nan_ptr[i] = 0; 
    block_num_inf_ptr[i] = 0; 
  }
}

template <typename T, typename MT>
__global__ void FindNanInfAndBlockMaxMin(T* value_ptr,
                                         const int64_t numel,
                                         int32_t* block_num_nan_ptr,
                                         int32_t* block_num_inf_ptr) {
  int64_t i = threadIdx.x + blockIdx.x * blockDim.x;

  int32_t num_nan = 0;
  int32_t num_inf = 0;

  for (; i < numel; i += blockDim.x * gridDim.x) {
    MT value = static_cast<MT>(value_ptr[i]);

    if (isnan(value)) {
      num_nan += 1;
    } else if (isinf(value)) {
      num_inf += 1;
    }
  }
  int32_t block_reduce_sum_nan = blockReduceSum(num_nan); 
  int32_t block_reduce_sum_inf = blockReduceSum(num_inf); 
  if(threadIdx.x == 0){
    atomicAdd(block_num_nan_ptr, block_reduce_sum_nan); 
    atomicAdd(block_num_inf_ptr, block_reduce_sum_inf); 
  }
}